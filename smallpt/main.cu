#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>   // smallpt, a Path Tracer by Kevin Beason, 2008
#include <stdlib.h> // Make : g++ -O3 -fopenmp smallpt.cpp -o smallpt
#include <stdio.h>  //        Remove "-fopenmp" for g++ version < 4.2
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "common/fmt.hpp"
#include "common/utils.hpp"

struct Vec {        // Usage: time ./smallpt 5000 && xv image.ppm
  double x, y, z;                  // position, also color (r,g,b)
  __host__ __device__ Vec(double x_=0, double y_=0, double z_=0){ x=x_; y=y_; z=z_; }
  __host__ __device__ Vec operator+(const Vec &b) const { return Vec(x+b.x,y+b.y,z+b.z); }
  __host__ __device__ Vec operator-(const Vec &b) const { return Vec(x-b.x,y-b.y,z-b.z); }
  __host__ __device__ Vec operator*(double b) const { return Vec(x*b,y*b,z*b); }
  __host__ __device__ Vec mult(const Vec &b) const { return Vec(x*b.x,y*b.y,z*b.z); }
  __host__ __device__ Vec& norm(){ return *this = *this * (1/sqrt(x*x+y*y+z*z)); }
  __host__ __device__ double dot(const Vec &b) const { return x*b.x+y*b.y+z*b.z; } // cross:
  __host__ __device__ Vec operator%(Vec&b){return Vec(y*b.z-z*b.y,z*b.x-x*b.z,x*b.y-y*b.x);}
};
struct Ray { Vec o, d; __host__ __device__ Ray(Vec o_, Vec d_) : o(o_), d(d_) {} };
enum Refl_t { DIFF, SPEC, REFR };  // material types, used in radiance(num_spheres, )
struct Sphere {
  double rad;       // radius
  Vec p, e, c;      // position, emission, color
  Refl_t refl;      // reflection type (DIFFuse, SPECular, REFRactive)
  __host__ __device__ Sphere(double rad_, Vec p_, Vec e_, Vec c_, Refl_t refl_):
                    rad(rad_), p(p_), e(e_), c(c_), refl(refl_) {}
  __host__ __device__ double intersect(const Ray &r) const { // returns distance, 0 if nohit
    Vec op = p-r.o; // Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0
    double t, eps=1e-4, b=op.dot(r.d), det=b*b-op.dot(op)+rad*rad;
    if (det<0) return 0; else det=sqrt(det);
    return (t=b-det)>eps ? t : ((t=b+det)>eps ? t : 0);
  }
};

__constant__ int w = 1024;
__constant__ int h = 768;
Sphere h_spheres[] = {//Scene: radius, position, emission, color, material
  Sphere(1e5, Vec( 1e5+1,40.8,81.6), Vec(),Vec(.75,.25,.25),DIFF),//Left
  Sphere(1e5, Vec(-1e5+99,40.8,81.6),Vec(),Vec(.25,.25,.75),DIFF),//Rght
  Sphere(1e5, Vec(50,40.8, 1e5),     Vec(),Vec(.75,.75,.75),DIFF),//Back
  Sphere(1e5, Vec(50,40.8,-1e5+170), Vec(),Vec(),           DIFF),//Frnt
  Sphere(1e5, Vec(50, 1e5, 81.6),    Vec(),Vec(.75,.75,.75),DIFF),//Botm
  Sphere(1e5, Vec(50,-1e5+81.6,81.6),Vec(),Vec(.75,.75,.75),DIFF),//Top
  Sphere(16.5,Vec(27,16.5,47),       Vec(),Vec(1,1,1)*.999, SPEC),//Mirr
  Sphere(16.5,Vec(73,16.5,78),       Vec(),Vec(1,1,1)*.999, REFR),//Glas
  Sphere(600, Vec(50,681.6-.27,81.6),Vec(12,12,12),  Vec(), DIFF) //Lite
};
__host__ __device__ inline double clamp(double x){ return x<0 ? 0 : x>1 ? 1 : x; }
__host__ __device__ inline int toInt(double x){ return int(pow(clamp(x),1/2.2)*255+.5); }
__device__ inline bool intersect(const int num_spheres, const Sphere* spheres, const Ray &r, double &t, int &id){
  double n=num_spheres, d, inf=t=1e20;
  for(int i=int(n);i--;) if((d=spheres[i].intersect(r))&&d<t){t=d;id=i;}
  return t<inf;
}
__device__ Vec radiance(const int num_spheres, const Sphere* spheres, Ray r, int depth, unsigned short *Xi, hiprandState* state){
  double t;                               // distance to intersection
  int id=0;                               // id of intersected object

  Vec s_e[10];
  Vec s_f[10];
  for(int i=0; i<10; i++){
    if (!intersect(num_spheres, spheres, r, t, id)) return Vec(); // if miss, return black
    const Sphere &obj = spheres[id];        // the hit object
    Vec x=r.o+r.d*t, n=(x-obj.p).norm(), nl=n.dot(r.d)<0?n:n*-1, f=obj.c;
    double p = f.x>f.y && f.x>f.z ? f.x : f.y>f.z ? f.y : f.z; // max refl
    if (i>5) {
      if (hiprand_uniform_double(state)<p) f=f*(1/p); 
      else {
        Vec res = obj.e;
        Vec e,f;
        while(--i>=0){
          e = s_e[i];
          f = s_f[i];
          res = e + f.mult(res);
        }
        return res;
      }
    } //return obj.e; //R.R.
    s_e[i]=obj.e;
    s_f[i]=f;
    if (obj.refl == DIFF){                  // Ideal DIFFUSE reflection
      double r1=2*M_PI*hiprand_uniform_double(state), r2=hiprand_uniform_double(state), r2s=sqrt(r2);
      Vec w=nl, u=((fabs(w.x)>.1?Vec(0,1):Vec(1))%w).norm(), v=w%u;
      Vec d = (u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrt(1-r2)).norm();
      r = Ray(x,d);
      continue;
      //return obj.e + f.mult(radiance(num_spheres, spheres,Ray(x,d),depth,Xi,state));
    } else if (obj.refl == SPEC) {r = Ray(x,r.d-n*2*n.dot(r.d)); continue;}
    Ray reflRay(x, r.d-n*2*n.dot(r.d));     // Ideal dielectric REFRACTION
    bool into = n.dot(nl)>0;                // Ray from outside going in?
    double nc=1, nt=1.5, nnt=into?nc/nt:nt/nc, ddn=r.d.dot(nl), cos2t;
    if ((cos2t=1-nnt*nnt*(1-ddn*ddn))<0) {r=reflRay; continue;}   // Total internal reflection
      //return obj.e + f.mult(radiance(num_spheres, spheres,reflRay,depth,Xi,state)); 
    Vec tdir = (r.d*nnt - n*((into?1:-1)*(ddn*nnt+sqrt(cos2t)))).norm();
    double a=nt-nc, b=nt+nc, R0=a*a/(b*b), c = 1-(into?-ddn:tdir.dot(n));
    double Re=R0+(1-R0)*c*c*c*c*c,Tr=1-Re,P=.25+.5*Re,RP=Re/P,TP=Tr/(1-P);

    if(hiprand_uniform_double(state)<P){
      r = reflRay;
      s_e[i] = s_e[i] * RP;
      s_f[i] = s_f[i] * RP;
    }
    else {
      r = Ray(x,tdir);
      s_e[i] = s_e[i] * TP;
      s_f[i] = s_f[i] * TP;
     }
    
  }
  return Vec();
}

__global__ void render(const int num_spheres, const Sphere* spheres, Vec* c, int samps){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  Ray cam(Vec(50,52,295.6), Vec(0,-0.042612,-1).norm()); // cam pos, dir
  Vec cx=Vec(w*.5135/h), cy=(cx%cam.d).norm()*.5135, r;

  if (y<h && x<w){ 
    unsigned short Xi[3]={0,0,y*y*y};
    hiprandState state;
    hiprand_init(y*y*y, 0, 0, &state);
 
    for (int sy=0, i=(h-y-1)*w+x; sy<2; sy++){ 
        for (int sx=0; sx<2; sx++, r=Vec()){        // 2x2 subpixel cols
          for (int s=0; s<samps; s++){
            double r1=2*hiprand_uniform_double(&state), dx=r1<1 ? sqrt(r1)-1: 1-sqrt(2-r1);
            double r2=2*hiprand_uniform_double(&state), dy=r2<1 ? sqrt(r2)-1: 1-sqrt(2-r2);
            Vec d = cx*( ( (sx+.5 + dx)/2 + x)/w - .5) +
                    cy*( ( (sy+.5 + dy)/2 + y)/h - .5) + cam.d;
            r = r + radiance(num_spheres, spheres, Ray(cam.o+d*140,d.norm()),0,Xi,&state)*(1./samps);
          } // Camera rays are pushed ^^^^^ forward to start in interior
          c[i] = c[i] + Vec(clamp(r.x),clamp(r.y),clamp(r.z))*.25;
        }
      }
  }
}
int main(int argc, char *argv[]){
  int w=1024, h=768, samps = argc==2 ? atoi(argv[1])/4 : 1; // # samples

  timer_start("Getting GPU Data."); //@@ start a timer
  // CUDA memory allocation
  Sphere *spheres;
  Vec *h_c=new Vec[w*h];
  Vec *c;
  int num_spheres = sizeof(h_spheres)/sizeof(Sphere);
  
  hipMalloc((void **)&c, w*h*sizeof(Vec));
  hipMalloc((void **)&spheres, sizeof(h_spheres));

  hipMemset(c, 0, w*h*sizeof(Vec));
  hipMemcpy(spheres, &h_spheres, sizeof(h_spheres), hipMemcpyHostToDevice);

  dim3 dimGrid((w-1)/16+1, (h-1)/16+1, 1);
  dim3 dimBlock(16, 16, 1);
  
  render<<<dimGrid, dimBlock>>>(num_spheres, spheres, c, samps);
  hipMemcpy(h_c, c, w*h*sizeof(Vec), hipMemcpyDeviceToHost);

  hipFree(c);
  hipFree(spheres);

  timer_stop(); //@@ stop the timer

  FILE *f = fopen("image-cuda.ppm", "w");         // Write image to PPM file.
  fprintf(f, "P3\n%d %d\n%d\n", w, h, 255);
  for (int i=0; i<w*h; i++)
    fprintf(f,"%d %d %d ", toInt(h_c[i].x), toInt(h_c[i].y), toInt(h_c[i].z));
}
